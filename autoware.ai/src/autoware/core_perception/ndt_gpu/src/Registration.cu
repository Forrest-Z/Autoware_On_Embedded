#include "hip/hip_runtime.h"
#include "ndt_gpu/Registration.h"
#include "ndt_gpu/debug.h"
#include <iostream>

namespace gpu {

GRegistration::GRegistration()
{
	max_iterations_ = 0;
	x_ = y_ = z_ = NULL;
	points_number_ = 0;

	trans_x_ = trans_y_ = trans_z_ = NULL;

	converged_ = false;
	nr_iterations_ = 0;

	transformation_epsilon_ = 0;
	target_cloud_updated_ = true;
	target_points_number_ = 0;

	target_x_ = target_y_ = target_z_ = NULL;
	is_copied_ = false;	

	if(GPU_PROFILING == 1){
		hipEventCreate(&event_start);
		hipEventCreate(&event_stop);
	}

}

GRegistration::GRegistration(const GRegistration &other)
{
	transformation_epsilon_ = other.transformation_epsilon_;
	max_iterations_ = other.max_iterations_;

	//Original scanned point clouds
	x_ = other.x_;
	y_ = other.y_;
	z_ = other.z_;

	points_number_ = other.points_number_;

	trans_x_ = other.trans_x_;
	trans_y_ = other.trans_y_;
	trans_z_ = other.trans_z_;

	converged_ = other.converged_;

	nr_iterations_ = other.nr_iterations_;
	final_transformation_ = other.final_transformation_;
	transformation_ = other.transformation_;
	previous_transformation_ = other.previous_transformation_;

	target_cloud_updated_ = other.target_cloud_updated_;

	target_x_ = other.target_x_;
	target_y_ = other.target_y_;
	target_z_ = other.target_z_;

	target_points_number_ = other.target_points_number_;
	is_copied_ = true;
}

GRegistration::~GRegistration()
{
	if (!is_copied_) {
		if (x_ != NULL) {
			checkCudaErrors(hipFree(x_));
			x_ = NULL;
		}

		if (y_ != NULL) {
			checkCudaErrors(hipFree(y_));
			y_ = NULL;
		}

		if (z_ != NULL) {
			checkCudaErrors(hipFree(z_));
			z_ = NULL;
		}

		if (trans_x_ != NULL) {
			checkCudaErrors(hipFree(trans_x_));
			trans_x_ = NULL;
		}

		if (trans_y_ != NULL) {
			checkCudaErrors(hipFree(trans_y_));
			trans_y_ = NULL;
		}

		if (trans_z_ != NULL) {
			checkCudaErrors(hipFree(trans_z_));
			trans_z_ = NULL;
		}

		if (target_x_ != NULL) {
				checkCudaErrors(hipFree(target_x_));
			target_x_ = NULL;
		}

		if (target_y_ != NULL) {
			checkCudaErrors(hipFree(target_y_));
			target_y_ = NULL;
		}

		if (target_z_ != NULL) {
			checkCudaErrors(hipFree(target_z_));
			target_z_ = NULL;
		}
	}
}

void GRegistration::setTransformationEpsilon(double trans_eps)
{
	transformation_epsilon_ = trans_eps;
}

double GRegistration::getTransformationEpsilon() const
{
	return transformation_epsilon_;
}

void GRegistration::setMaximumIterations(int max_itr)
{
	max_iterations_ = max_itr;
}

int GRegistration::getMaximumIterations() const
{
	return max_iterations_;
}

Eigen::Matrix<float, 4, 4> GRegistration::getFinalTransformation() const
{
	return final_transformation_;
}

int GRegistration::getFinalNumIteration() const
{
	return nr_iterations_;
}

bool GRegistration::hasConverged() const
{
	return converged_;
}


template <typename T>
__global__ void convertInput(T *input, float *out_x, float *out_y, float *out_z, int point_num)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	for (int i = idx; i < point_num; i += stride) {
		T tmp = input[i];
		out_x[i] = tmp.x;
		out_y[i] = tmp.y;
		out_z[i] = tmp.z;
	}
}

void GRegistration::setInputSource(pcl::PointCloud<pcl::PointXYZI>::Ptr input)
{
	//Convert point cloud to float x, y, z
	if (input->size() > 0) {
		points_number_ = input->size();

		pcl::PointXYZI *tmp;

		checkCudaErrors(hipMalloc(&tmp, sizeof(pcl::PointXYZI) * points_number_));

		pcl::PointXYZI *host_tmp = input->points.data();

		// Pin the host buffer for accelerating the memory copy
#ifndef __aarch64__
		checkCudaErrors(hipHostRegister(host_tmp, sizeof(pcl::PointXYZI) * points_number_, hipHostRegisterDefault));
#endif
		start_profiling();
		checkCudaErrors(hipMemcpy(tmp, host_tmp, sizeof(pcl::PointXYZI) * points_number_, hipMemcpyHostToDevice));
		stop_profiling(HTOD);

		if (x_ != NULL) {
			checkCudaErrors(hipFree(x_));
			x_ = NULL;
		}

		if (y_ != NULL) {
			checkCudaErrors(hipFree(y_));
			y_ = NULL;
		}

		if (z_ != NULL) {
			checkCudaErrors(hipFree(z_));
			z_ = NULL;
		}

		checkCudaErrors(hipMalloc(&x_, sizeof(float) * points_number_));
		checkCudaErrors(hipMalloc(&y_, sizeof(float) * points_number_));
		checkCudaErrors(hipMalloc(&z_, sizeof(float) * points_number_));

		int block_x = (points_number_ > BLOCK_SIZE_X) ? BLOCK_SIZE_X : points_number_;
		int grid_x = (points_number_ - 1) / block_x + 1;

		start_profiling();
		convertInput<pcl::PointXYZI><<<grid_x, block_x>>>(tmp, x_, y_, z_, points_number_);
		checkCudaErrors(hipGetLastError());
		checkCudaErrors(hipDeviceSynchronize());
		stop_profiling(LAUNCH);


		if (trans_x_ != NULL) {
			checkCudaErrors(hipFree(trans_x_));
			trans_x_ = NULL;
		}

		if (trans_y_ != NULL) {
			checkCudaErrors(hipFree(trans_y_));
			trans_y_ = NULL;
		}

		if (trans_z_ != NULL) {
			checkCudaErrors(hipFree(trans_z_));
			trans_z_ = NULL;
		}

		checkCudaErrors(hipMalloc(&trans_x_, sizeof(float) * points_number_));
		checkCudaErrors(hipMalloc(&trans_y_, sizeof(float) * points_number_));
		checkCudaErrors(hipMalloc(&trans_z_, sizeof(float) * points_number_));

		// Initially, also copy scanned points to transformed buffers
		start_profiling();
		checkCudaErrors(hipMemcpy(trans_x_, x_, sizeof(float) * points_number_, hipMemcpyDeviceToDevice));
		stop_profiling(DTOH);

		start_profiling();
		checkCudaErrors(hipMemcpy(trans_y_, y_, sizeof(float) * points_number_, hipMemcpyDeviceToDevice));
		stop_profiling(DTOH);

		start_profiling();
		checkCudaErrors(hipMemcpy(trans_z_, z_, sizeof(float) * points_number_, hipMemcpyDeviceToDevice));
		stop_profiling(DTOH);

		checkCudaErrors(hipFree(tmp));

		// Unpin host buffer
#ifndef __aarch64__
		checkCudaErrors(hipHostUnregister(host_tmp));
#endif
	}
}

void GRegistration::setInputSource(pcl::PointCloud<pcl::PointXYZ>::Ptr input)
{
	//Convert point cloud to float x, y, z
	if (input->size() > 0) {
		points_number_ = input->size();

		pcl::PointXYZ *tmp;

		checkCudaErrors(hipMalloc(&tmp, sizeof(pcl::PointXYZ) * points_number_));

		pcl::PointXYZ *host_tmp = input->points.data();

		// Pin the host buffer for accelerating the memory copy
#ifndef __aarch64__
		checkCudaErrors(hipHostRegister(host_tmp, sizeof(pcl::PointXYZ) * points_number_, hipHostRegisterDefault));
#endif	
		start_profiling();
		checkCudaErrors(hipMemcpy(tmp, host_tmp, sizeof(pcl::PointXYZ) * points_number_, hipMemcpyHostToDevice));
		stop_profiling(HTOD);

		if (x_ != NULL) {
			checkCudaErrors(hipFree(x_));
			x_ = NULL;
		}

		if (y_ != NULL) {
			checkCudaErrors(hipFree(y_));
			y_ = NULL;
		}

		if (z_ != NULL) {
			checkCudaErrors(hipFree(z_));
			z_ = NULL;
		}

		checkCudaErrors(hipMalloc(&x_, sizeof(float) * points_number_));
		checkCudaErrors(hipMalloc(&y_, sizeof(float) * points_number_));
		checkCudaErrors(hipMalloc(&z_, sizeof(float) * points_number_));

		int block_x = (points_number_ > BLOCK_SIZE_X) ? BLOCK_SIZE_X : points_number_;
		int grid_x = (points_number_ - 1) / block_x + 1;

		start_profiling();
		convertInput<pcl::PointXYZ><<<grid_x, block_x>>>(tmp, x_, y_, z_, points_number_);
		checkCudaErrors(hipGetLastError());
		checkCudaErrors(hipDeviceSynchronize());
		stop_profiling(LAUNCH);

		if (trans_x_ != NULL) {
			checkCudaErrors(hipFree(trans_x_));
			trans_x_ = NULL;
		}

		if (trans_y_ != NULL) {
			checkCudaErrors(hipFree(trans_y_));
			trans_y_ = NULL;
		}

		if (trans_z_ != NULL) {
			checkCudaErrors(hipFree(trans_z_));
			trans_z_ = NULL;
		}

		checkCudaErrors(hipMalloc(&trans_x_, sizeof(float) * points_number_));
		checkCudaErrors(hipMalloc(&trans_y_, sizeof(float) * points_number_));
		checkCudaErrors(hipMalloc(&trans_z_, sizeof(float) * points_number_));

		start_profiling();
		checkCudaErrors(hipMemcpy(trans_x_, x_, sizeof(float) * points_number_, hipMemcpyDeviceToDevice));		
		stop_profiling(DTOH);

		start_profiling();
		checkCudaErrors(hipMemcpy(trans_y_, y_, sizeof(float) * points_number_, hipMemcpyDeviceToDevice));
		stop_profiling(DTOH);

		start_profiling();
		checkCudaErrors(hipMemcpy(trans_z_, z_, sizeof(float) * points_number_, hipMemcpyDeviceToDevice));
		stop_profiling(DTOH);
		

		checkCudaErrors(hipFree(tmp));
#ifndef __aarch64__
		checkCudaErrors(hipHostUnregister(host_tmp));
#endif
	}
}



//Set input MAP data
void GRegistration::setInputTarget(pcl::PointCloud<pcl::PointXYZI>::Ptr input)
{
	if (input->size() > 0) {
		target_points_number_ = input->size();

		pcl::PointXYZI *tmp;

		checkCudaErrors(hipMalloc(&tmp, sizeof(pcl::PointXYZI) * target_points_number_));

		pcl::PointXYZI *host_tmp = input->points.data();

#ifndef __aarch64__
		checkCudaErrors(hipHostRegister(host_tmp, sizeof(pcl::PointXYZI) * target_points_number_, hipHostRegisterDefault));
#endif

		start_profiling();
		checkCudaErrors(hipMemcpy(tmp, host_tmp, sizeof(pcl::PointXYZI) * target_points_number_, hipMemcpyHostToDevice));
		stop_profiling(HTOD);

		if (target_x_ != NULL) {
			checkCudaErrors(hipFree(target_x_));
			target_x_ = NULL;
		}

		if (target_y_ != NULL) {
			checkCudaErrors(hipFree(target_y_));
			target_y_ = NULL;
		}

		if (target_z_ != NULL) {
			checkCudaErrors(hipFree(target_z_));
			target_z_ = NULL;
		}

		checkCudaErrors(hipMalloc(&target_x_, sizeof(float) * target_points_number_));
		checkCudaErrors(hipMalloc(&target_y_, sizeof(float) * target_points_number_));
		checkCudaErrors(hipMalloc(&target_z_, sizeof(float) * target_points_number_));

		int block_x = (target_points_number_ > BLOCK_SIZE_X) ? BLOCK_SIZE_X : target_points_number_;
		int grid_x = (target_points_number_ - 1) / block_x + 1;

		start_profiling();
		convertInput<pcl::PointXYZI><<<grid_x, block_x>>>(tmp, target_x_, target_y_, target_z_, target_points_number_);
		checkCudaErrors(hipGetLastError());
		checkCudaErrors(hipDeviceSynchronize());
		stop_profiling(LAUNCH);

#ifndef __aarch64__
		checkCudaErrors(hipHostUnregister(host_tmp));
#endif
		checkCudaErrors(hipFree(tmp));
	}
}

void GRegistration::setInputTarget(pcl::PointCloud<pcl::PointXYZ>::Ptr input)
{
	if (input->size() > 0) {
		target_points_number_ = input->size();

		pcl::PointXYZ *tmp;

		checkCudaErrors(hipMalloc(&tmp, sizeof(pcl::PointXYZ) * target_points_number_));

		pcl::PointXYZ *host_tmp = input->points.data();

#ifndef __aarch64__
		checkCudaErrors(hipHostRegister(host_tmp, sizeof(pcl::PointXYZ) * target_points_number_, hipHostRegisterDefault));
#endif
		start_profiling();
		checkCudaErrors(hipMemcpy(tmp, host_tmp, sizeof(pcl::PointXYZ) * target_points_number_, hipMemcpyHostToDevice));
		stop_profiling(HTOD);

		if (target_x_ != NULL) {
			checkCudaErrors(hipFree(target_x_));
			target_x_ = NULL;
		}

		if (target_y_ != NULL) {
			checkCudaErrors(hipFree(target_y_));
			target_y_ = NULL;
		}

		if (target_z_ != NULL) {
			checkCudaErrors(hipFree(target_z_));
			target_z_ = NULL;
		}

		checkCudaErrors(hipMalloc(&target_x_, sizeof(float) * target_points_number_));
		checkCudaErrors(hipMalloc(&target_y_, sizeof(float) * target_points_number_));
		checkCudaErrors(hipMalloc(&target_z_, sizeof(float) * target_points_number_));

		int block_x = (target_points_number_ > BLOCK_SIZE_X) ? BLOCK_SIZE_X : target_points_number_;
		int grid_x = (target_points_number_ - 1) / block_x + 1;

		start_profiling();
		convertInput<pcl::PointXYZ><<<grid_x, block_x>>>(tmp, target_x_, target_y_, target_z_, target_points_number_);		
		checkCudaErrors(hipGetLastError());
		checkCudaErrors(hipDeviceSynchronize());
		stop_profiling(LAUNCH);

		checkCudaErrors(hipFree(tmp));
#ifndef __aarch64__
		checkCudaErrors(hipHostUnregister(host_tmp));
#endif
	}
}

void GRegistration::align(const Eigen::Matrix<float, 4, 4> &guess)
{
	converged_ = false;

	final_transformation_ = transformation_ = previous_transformation_ = Eigen::Matrix<float, 4, 4>::Identity();

	computeTransformation(guess);
}

void GRegistration::computeTransformation(const Eigen::Matrix<float, 4, 4> &guess) {
	printf("Unsupported by Registration\n");
}

void GRegistration::start_profiling(){
	if(GPU_PROFILING == 1)
		hipEventRecord(event_start, 0);
}

void GRegistration::stop_profiling(int type){
	if(GPU_PROFILING == 1){		
		float time;
		hipEventRecord(event_stop, 0);
		hipEventSynchronize(event_stop);
		hipEventElapsedTime(&time, event_start, event_stop);
		write_data(gid, time, type);
		gid++;
	}
}

void GRegistration::write_data(int id, float time, int type){
	if(GPU_PROFILING == 1){
		fprintf(fp, "%d, %f, %d\n", id, time, type);				
	}
}

void GRegistration::write_dummy_line(){
	if(GPU_PROFILING == 1){
		fprintf(fp, "-1, -1, -1\n");						
		fflush(fp);
		gid = 0;
	}
}

void GRegistration::initialize_file(const char name[]){
	if(GPU_PROFILING == 1){
		fp = fopen(name, "w+");
		fprintf(fp, "ID, TIME, TYPE\n");		
	}
}



void GRegistration::close_file(){
	if(GPU_PROFILING == 1)
		fclose(fp);
}

}
