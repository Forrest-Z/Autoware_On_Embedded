#include "ndt_gpu/MatrixDevice.h"
#include "ndt_gpu/debug.h"
#include "rubis_sched/sched.hpp"

namespace gpu {
MatrixDevice::MatrixDevice(int rows, int cols) {
	rows_ = rows;
	cols_ = cols;
	offset_ = 1;
	fr_ = true;
	buffer_ = NULL;
	memAllocId = 0;
	memFreeId = 0;
}

void MatrixDevice::memAlloc()
{
	if (buffer_ != NULL && fr_) {
		//rubis::sched::request_gpu(1);
		checkCudaErrors(hipFree(buffer_));
		//rubis::sched::yield_gpu(1,"free");
		buffer_ = NULL;
	}

	//rubis::sched::request_gpu(2);
	checkCudaErrors(hipMalloc(&buffer_, sizeof(double) * rows_ * cols_ * offset_));
	//rubis::sched::yield_gpu(2,"hipMalloc");

	//rubis::sched::request_gpu(3);
	checkCudaErrors(hipMemset(buffer_, 0, sizeof(double) * rows_ * cols_ * offset_));
	//rubis::sched::yield_gpu(3,"hipMemset");

	checkCudaErrors(hipDeviceSynchronize());
	fr_ = true;
}

void MatrixDevice::memAlloc_free()
{
	if (buffer_ != NULL && fr_) {
		//rubis::sched::request_gpu(4);
		checkCudaErrors(hipFree(buffer_));
		//rubis::sched::yield_gpu(4,"free");
		buffer_ = NULL;
	}
}

void MatrixDevice::memAlloc_malloc()
{
	//rubis::sched::request_gpu(5);
	checkCudaErrors(hipMalloc(&buffer_, sizeof(double) * rows_ * cols_ * offset_));
	//rubis::sched::yield_gpu(5,"hipMalloc");
}

void MatrixDevice::memAlloc_memset()
{
	//rubis::sched::request_gpu(6);
	checkCudaErrors(hipMemset(buffer_, 0, sizeof(double) * rows_ * cols_ * offset_));
	//rubis::sched::yield_gpu(6,"hipMemset");

	fr_ = true;
}

void MatrixDevice::memFree()
{
	if (fr_) {
		if (buffer_ != NULL) {
			//rubis::sched::request_gpu(7);
			checkCudaErrors(hipFree(buffer_));
			//rubis::sched::yield_gpu(7,"free");
			buffer_ = NULL;
		}
	}
}


SquareMatrixDevice::SquareMatrixDevice(int size) :
	MatrixDevice(size, size)
{

}

}
